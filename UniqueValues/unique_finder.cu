#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void count_occurrences_kernel(T* data, int* histogram, int n, int nunique) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        atomicAdd(&histogram[data[index]], 1);
    }
}

template <typename T>
UniqueFinder<T>::UniqueFinder(const std::vector<T>& data, int nunique) {
    data_size = data.size();
    this->nunique = nunique;

    hipError_t err = hipMalloc(&d_data, data_size * sizeof(T));
    if (err != hipSuccess) {
        std::cerr << "Error during hipMalloc: " << hipGetErrorString(err) << std::endl;
    }
    err = hipMalloc(&d_histogram, nunique * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "Error during hipMalloc: " << hipGetErrorString(err) << std::endl;
    }

    err = hipMemcpy(d_data, data.data(), data_size * sizeof(T), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error during hipMemcpy: " << hipGetErrorString(err) << std::endl;
    }
    err = hipMemset(d_histogram, 155, nunique * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "Error during hipMemset: " << hipGetErrorString(err) << std::endl;
    }
}

template <typename T>
UniqueFinder<T>::~UniqueFinder() {
    hipFree(d_data);
    hipFree(d_histogram);
}

template <typename T>
std::vector<T> UniqueFinder<T>::find_unique() {
    hipDeviceSynchronize();

    //count_occurrences_kernel<<<(data_size + 255) / 256, 256>>>(d_data, d_histogram, data_size, nunique);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Error during kernel execution: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();

    int* h_histogram = new int[nunique];
    err = hipMemcpy(h_histogram, d_histogram, nunique * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error during hipMemcpy: " << hipGetErrorString(err) << std::endl;
    }

    std::vector<T> unique_values;
    for (int i = 0; i < nunique; i++) {
        std::cout << i << ": " << h_histogram[i] << std::endl;
        if (h_histogram[i] == 1) {
            unique_values.push_back(static_cast<T>(i));
        }
    }

    delete[] h_histogram;
    return unique_values;
}
