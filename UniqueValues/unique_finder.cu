#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void count_occurrences_kernel(int* data, int* histogram, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        atomicAdd(&histogram[data[index]], 1);
    }
}

UniqueFinder::UniqueFinder(const std::vector<int>& data, int nunique) {
    this->data = data;
    this->unique_values = nunique;
}

UniqueFinder::~UniqueFinder() {
}

std::vector<int> UniqueFinder::find_unique() {
    int n = this->data.size();
    int nunique = this->unique_values;

    int* d_data;
    int* d_histogram;
    
    hipMalloc(&d_data, n * sizeof(int));
    hipMalloc(&d_histogram, nunique * sizeof(int));
    hipMemcpy(d_data, data.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, nunique * sizeof(int));

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    count_occurrences_kernel<<<blocks, threadsPerBlock>>>(d_data, d_histogram, n);

    int* h_histogram = new int[nunique];
    hipMemcpy(h_histogram, d_histogram, nunique * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<int> unique_elements;
    for (int i = 0; i < nunique; i++) {
        if (h_histogram[i] == 1) {
            unique_elements.push_back(i);
            std::cout << i << " ";
        }
    }

    hipFree(d_data);
    hipFree(d_histogram);
    delete[] h_histogram;
}
