#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

enum { BLOCK_SIZE = 1024, CHUNK_SIZE = 4096 };

__global__ void count_occurrences_kernel(int* data, int* global_histogram, int n, int nunique, int chunk_size) {
    extern __shared__ int local_histogram[];

    int threadId = threadIdx.x;
    int globalId = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize local histogram in shared memory
    for (int i = threadId; i < nunique; i += blockDim.x) {
        local_histogram[i] = 0;
    }
    __syncthreads();

    // Each thread processes a chunk of data and updates the local histogram
    for (int i = 0; i < chunk_size; i++) {
        int dataIdx = globalId * chunk_size + i;
        if (dataIdx < n) {
            atomicAdd(&local_histogram[data[dataIdx]], 1);
        }
    }
    __syncthreads();

    // Update global histogram from local histograms
    for (int i = threadId; i < nunique; i += blockDim.x) {
        atomicAdd(&global_histogram[i], local_histogram[i]);
    }
}

__global__ void histogram_to_binary(int* histogram, int* binary, int nunique) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nunique) {
        binary[index] = (histogram[index] == 1) ? 1 : 0;
    }
}

__global__ void simple_prefix_sum(int* input, int* output, int n) {
    extern __shared__ int temp[];

    int threadId = threadIdx.x;

    // Load input into shared memory.
    temp[threadId] = (threadId < n) ? input[threadId] : 0;
    __syncthreads();

    for (int stride = 1; stride < n; stride *= 2) {
        int value = 0;
        if (threadId >= stride) {
            value = temp[threadId - stride];
        }
        __syncthreads();
        temp[threadId] += value;
        __syncthreads();
    }

    if (threadId < n) {
        output[threadId] = temp[threadId];
    }
}

__global__ void extract_unique_values(int* histogram, int* prefixSum, int* unique_values, int nunique) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nunique) {
        if (histogram[index] == 1) {
            unique_values[prefixSum[index] - 1] = index;
        }
    }
}

UniqueFinder::UniqueFinder(const std::vector<int>& data, int nunique) {
    this->data = data;
    this->unique_values = nunique;
}

UniqueFinder::~UniqueFinder() {
}

std::vector<int> UniqueFinder::find_unique() {
    int n = this->data.size();
    int nunique = this->unique_values;

    int* d_data;
    int* d_histogram;
    
    hipMalloc(&d_data, n * sizeof(int));
    hipMalloc(&d_histogram, nunique * sizeof(int));
    hipMemcpy(d_data, data.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, nunique * sizeof(int));

    // Obtain the histogram of the data
    int blocks_count = (n + CHUNK_SIZE - 1) / CHUNK_SIZE;
    count_occurrences_kernel<<<blocks_count, BLOCK_SIZE, nunique * sizeof(int)>>>(d_data, d_histogram, n, nunique, CHUNK_SIZE);

    // Convert histogram to binary format
    int* d_binary;
    hipMalloc(&d_binary, nunique * sizeof(int));
    histogram_to_binary<<<(nunique + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_histogram, d_binary, nunique);

    // Allocate memory for prefix_sum and unique_values on the device
    int* d_prefix_sum, *d_unique_values;
    hipMalloc(&d_prefix_sum, nunique * sizeof(int));
    hipMalloc(&d_unique_values, nunique * sizeof(int));

    // Compute prefix sum
    int blockSize = min(nunique, BLOCK_SIZE);
    simple_prefix_sum<<<1, blockSize, blockSize * sizeof(int)>>>(d_binary, d_prefix_sum, nunique);

    // Extract unique values based on the prefix sum
    extract_unique_values<<<(nunique + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_histogram, d_prefix_sum, d_unique_values, nunique);

    // Get the number of unique values
    int num_unique;
    hipMemcpy(&num_unique, &d_prefix_sum[nunique - 1], sizeof(int), hipMemcpyDeviceToHost);

    // Allocate space for these unique values on the host
    std::vector<int> unique_elements(num_unique);

    // Copy the unique values from the device to the host memory
    hipMemcpy(unique_elements.data(), d_unique_values, num_unique * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_data);
    hipFree(d_histogram);
    hipFree(d_prefix_sum);
    hipFree(d_binary);
    hipFree(d_unique_values);

    return unique_elements;
}
