#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void count_occurrences_kernel(T* data, int* histogram, size_t n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        atomicAdd(&histogram[data[index]], 1);
    }
}

template <typename T>
UniqueFinder<T>::UniqueFinder(const std::vector<T>& data, size_t nunique) {
    data_size = data.size();
    this->nunique = nunique;

    hipError_t err = hipMalloc(&d_data, data_size * sizeof(T));
    if (err != hipSuccess) {
        std::cerr << "Error during hipMalloc: " << hipGetErrorString(err) << std::endl;
    }
    err = hipMalloc(&d_histogram, nunique * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "Error during hipMalloc: " << hipGetErrorString(err) << std::endl;
    }

    err = hipMemcpy(d_data, data.data(), data_size * sizeof(T), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error during hipMemcpy: " << hipGetErrorString(err) << std::endl;
    }
    err = hipMemset(d_histogram, 0, nunique * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "Error during hipMemset: " << hipGetErrorString(err) << std::endl;
    }
}

template <typename T>
UniqueFinder<T>::~UniqueFinder() {
    hipFree(d_data);
    hipFree(d_histogram);
}

template <typename T>
std::vector<T> UniqueFinder<T>::find_unique() {
    err = count_occurrences_kernel<<<(data_size + 255) / 256, 256>>>(d_data, d_histogram, data_size);
    if (err != hipSuccess) {
        std::cerr << "Error during kernel execution: " << hipGetErrorString(err) << std::endl;
    }

    int* h_histogram = new int[nunique];
    err = hipMemcpy(h_histogram, d_histogram, nunique * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error during hipMemcpy: " << hipGetErrorString(err) << std::endl;
    }

    std::vector<T> unique_values;
    for (size_t i = 0; i < nunique; i++) {
        std::cout << i << ": " << h_histogram[i] << std::endl;
        if (h_histogram[i] == 1) {
            unique_values.push_back(static_cast<T>(i));
        }
    }

    delete[] h_histogram;
    return unique_values;
}
