#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

enum { BLOCK_SIZE = 256 };

__global__ void count_occurrences_kernel(int* data, int* histogram, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        atomicAdd(&histogram[data[index]], 1);
    }
}

UniqueFinder::UniqueFinder(const std::vector<int>& data, int nunique) {
    this->data = data;
    this->unique_values = nunique;
}

UniqueFinder::~UniqueFinder() {
}

std::vector<int> UniqueFinder::find_unique() {
    int n = this->data.size();
    int nunique = this->unique_values;

    int* d_data;
    int* d_histogram;
    
    hipMalloc(&d_data, n * sizeof(int));
    hipMalloc(&d_histogram, nunique * sizeof(int));
    hipMemcpy(d_data, data.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, nunique * sizeof(int));

    int blocks_count = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    count_occurrences_kernel<<<blocks, BLOCK_SIZE>>>(d_data, d_histogram, n);

    int* h_histogram = new int[nunique];
    hipMemcpy(h_histogram, d_histogram, nunique * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<int> unique_elements;
    for (int i = 0; i < nunique; i++) {
        if (h_histogram[i] == 1) {
            unique_elements.push_back(i);
        }
    }

    hipFree(d_data);
    hipFree(d_histogram);
    delete[] h_histogram;

    return unique_elements;
}
