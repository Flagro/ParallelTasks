#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

enum { BLOCK_SIZE = 256, CHUNK_SIZE = 512 };

__global__ void count_occurrences_kernel(int* data, int* global_histogram, int n, int nunique, int chunk_size) {
    extern __shared__ int local_histogram[];

    int threadId = threadIdx.x;
    int globalId = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize local histogram in shared memory
    for (int i = threadId; i < nunique; i += blockDim.x) {
        local_histogram[i] = 0;
    }
    __syncthreads();

    // Each thread processes a chunk of data and updates the local histogram
    for (int i = 0; i < chunk_size; i++) {
        int dataIdx = globalId * chunk_size + i;
        if (dataIdx < n) {
            atomicAdd(&local_histogram[data[dataIdx]], 1);
        }
    }
    __syncthreads();

    // Update global histogram from local histograms
    for (int i = threadId; i < nunique; i += blockDim.x) {
        atomicAdd(&global_histogram[i], local_histogram[i]);
    }
}

__global__ void histogram_to_binary(int* histogram, int* binary, int nunique) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nunique) {
        binary[index] = (histogram[index] == 1) ? 1 : 0;
    }
}

__global__ void prefix_sum_first_pass(int* input, int* output, int* blockSums, int n) {
    extern __shared__ int temp[];

    int threadId = threadIdx.x;
    int blockId = blockIdx.x;
    int offset = 1;

    int idx = 2 * threadId + blockId * 2 * blockDim.x;
    int idxNext = idx + 1;

    // Load input into shared memory
    temp[2 * threadId] = (idx < n) ? input[idx] : 0;
    temp[2 * threadId + 1] = (idxNext < n) ? input[idxNext] : 0;
    __syncthreads();

    // Up-sweep phase (reduce)
    for (int d = blockDim.x; d > 0; d >>= 1) {
        __syncthreads();
        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    // Clear the last element
    if (threadId == 0) {
        temp[2 * blockDim.x - 1] = 0;
    }

    // Down-sweep phase (post-reduce)
    for (int d = 1; d < 2 * blockDim.x; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    // Write results to output
    if (idx < n) {
        output[idx] = temp[2 * threadId];
    }
    if (idxNext < n) {
        output[idxNext] = temp[2 * threadId + 1];
    }

    // Write the block's sum to blockSums
    if (threadId == 0) {
        blockSums[blockId] = temp[2 * blockDim.x - 2] + ((blockId * 2 * blockDim.x + 2 * blockDim.x - 1) < n ? input[blockId * 2 * blockDim.x + 2 * blockDim.x - 1] : 0);
    }
}

__global__ void add_block_sums(int* input, int* blockSums, int n) {
    int globalId = threadIdx.x + blockIdx.x * blockDim.x;

    if (globalId < n) {
        input[globalId] += blockSums[blockIdx.x];
    }
}

__global__ void extract_unique_values(int* histogram, int* prefixSum, int* data, int* unique_values, int nunique) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nunique) {
        if (histogram[index] == 1) {
            unique_values[prefixSum[index] - 1] = data[index];
        }
    }
}

int nextPowerOf2(int n) {
    int count = 0;
    if (n && !(n & (n - 1))) {
        return n;
    }
    while(n != 0) {
        n >>= 1;
        count += 1;
    }
    return 1 << count;
}

UniqueFinder::UniqueFinder(const std::vector<int>& data, int nunique) {
    this->data = data;
    this->unique_values = nunique;
}

UniqueFinder::~UniqueFinder() {
}

void recursive_prefix_sum(int* d_input, int* d_output, int* d_blockSums, int length) {
    int numBlocks = (length + BLOCK_SIZE * 2 - 1) / (BLOCK_SIZE * 2);
    if (numBlocks > 1) {
        // If numBlocks is greater than BLOCK_SIZE, then we need another layer of block sums
        int* d_nextBlockSums;
        hipMalloc(&d_nextBlockSums, numBlocks * sizeof(int));
        recursive_prefix_sum(d_blockSums, d_blockSums, d_nextBlockSums, numBlocks);
        hipFree(d_nextBlockSums);
    }
    prefix_sum_first_pass<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE * 2 * sizeof(int)>>>(d_input, d_output, d_blockSums, length);
    hipDeviceSynchronize();
    add_block_sums<<<numBlocks, BLOCK_SIZE>>>(d_output, d_blockSums, length);
    hipDeviceSynchronize();
}

std::vector<int> UniqueFinder::find_unique() {
    int n = this->data.size();
    int nunique = this->unique_values;

    int* d_data;
    int* d_histogram;
    
    hipMalloc(&d_data, n * sizeof(int));
    hipMalloc(&d_histogram, nunique * sizeof(int));
    hipMemcpy(d_data, data.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, nunique * sizeof(int));

    // Obtain the histogram of the data
    int blocks_count = (n + CHUNK_SIZE - 1) / CHUNK_SIZE;
    count_occurrences_kernel<<<blocks_count, BLOCK_SIZE, nunique * sizeof(int)>>>(d_data, d_histogram, n, nunique, CHUNK_SIZE);

    // Convert histogram to binary format
    int* d_binary;
    hipMalloc(&d_binary, nunique * sizeof(int));
    histogram_to_binary<<<(nunique + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_histogram, d_binary, nunique);

    // Allocate memory for prefix_sum and unique_values on the device
    int* d_prefix_sum, *d_unique_values;
    hipMalloc(&d_prefix_sum, nunique * sizeof(int));
    hipMalloc(&d_unique_values, nunique * sizeof(int));

    // Compute prefix sum
    int numBlocks = (nunique + BLOCK_SIZE * 2 - 1) / (BLOCK_SIZE * 2);
    int* d_blockSums;
    hipMalloc(&d_blockSums, numBlocks * sizeof(int));
    recursive_prefix_sum(d_binary, d_prefix_sum, d_blockSums, nunique);
    hipFree(d_blockSums);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error after launching [add_block_sums]: %s\n", hipGetErrorString(err));
    }

    // After computing prefix sum
    int* h_prefix_sum_debug = new int[nunique];
    hipMemcpy(h_prefix_sum_debug, d_prefix_sum, nunique * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < nunique; i++) {
        std::cout << "PrefixSum[" << i << "]: " << h_prefix_sum_debug[i] << std::endl;
    }
    delete[] h_prefix_sum_debug;




    // Extract unique values based on the prefix sum
    extract_unique_values<<<(nunique + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_histogram, d_binary, d_data, d_unique_values, nunique);

    // 1. Get the number of unique values
    int num_unique;
    hipMemcpy(&num_unique, &d_binary[nunique - 1], sizeof(int), hipMemcpyDeviceToHost);

    // 2. Allocate space for these unique values on the host
    std::vector<int> unique_elements(num_unique);

    // 3. Copy the unique values from the device to the host memory
    hipMemcpy(unique_elements.data(), d_unique_values, num_unique * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_data);
    hipFree(d_histogram);
    hipFree(d_prefix_sum);
    hipFree(d_binary);
    hipFree(d_unique_values);
    // delete[] h_histogram;
    // delete[] h_unique_values;

    return unique_elements;
}
