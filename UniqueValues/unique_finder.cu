#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

template <typename T>
__global__ void count_occurrences_kernel(const T* data, size_t data_size, int* histogram, size_t nunique, T* unique_values) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < data_size; i += stride) {
        T value = data[i];
        if (value >= 0 && value < nunique) {
            atomicAdd(&histogram[value], 1);
        }
    }
}

template <typename T>
__global__ void find_unique_kernel(int* histogram, size_t nunique, T* unique_values, int* unique_counter) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < nunique && histogram[index] == 1) {
        int pos = atomicAdd(unique_counter, 1);
        unique_values[pos] = index;
    }
}

template <typename T>
UniqueFinder<T>::UniqueFinder(const std::vector<T>& data, size_t nunique)
    : data_size(data.size()), nunique(nunique) {

    // Allocate device memory for data, histogram, and unique values
    hipMalloc(&d_data, data_size * sizeof(T));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipMalloc(&d_histogram, nunique * sizeof(int));
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipMalloc(&d_unique_values, nunique * sizeof(T));
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipMalloc(&d_unique_counter, sizeof(int));
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    // Copy data to device
    hipMemcpy(d_data, data.data(), data_size * sizeof(T), hipMemcpyHostToDevice);
    err = hipMemcpy(d_data, data.data(), data_size * sizeof(T), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error copying data to device: " << hipGetErrorString(err) << std::endl;
        // Handle the error, e.g., by exiting or throwing an exception
        exit(1);
    }

    // Initialize histogram and unique counter to zero
    hipMemset(d_histogram, 0, nunique * sizeof(int));
    hipMemset(d_unique_counter, 0, sizeof(int));
}

template <typename T>
UniqueFinder<T>::~UniqueFinder() {
    // Free device memory
    hipFree(d_data);
    hipFree(d_histogram);
    hipFree(d_unique_values);
    hipFree(d_unique_counter);
}

template <typename T>
std::vector<T> UniqueFinder<T>::find_unique() {
    int num_blocks = (data_size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Launch kernel to count occurrences
    count_occurrences_kernel<<<num_blocks, BLOCK_SIZE>>>(d_data, data_size, d_histogram, nunique, d_unique_values);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    // Launch kernel to find unique numbers
    find_unique_kernel<<<num_blocks, BLOCK_SIZE>>>(d_histogram, nunique, d_unique_values, d_unique_counter);
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    // Copy unique values to host
    int unique_count;
    hipMemcpy(&unique_count, d_unique_counter, sizeof(int), hipMemcpyDeviceToHost);

    std::vector<T> unique_values(unique_count);
    hipMemcpy(unique_values.data(), d_unique_values, unique_count * sizeof(T), hipMemcpyDeviceToHost);

    return unique_values;
}
