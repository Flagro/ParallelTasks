#include "hip/hip_runtime.h"
#include "unique_finder.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

enum { BLOCK_SIZE = 1024, CHUNK_SIZE = 128000 };

/*
__global__ void count_occurrences_kernel(int* data, int* global_histogram, int n, int nunique, int chunk_size) {
    extern __shared__ int local_histogram[];

    int threadId = threadIdx.x;
    int globalId = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize local histogram in shared memory
    for (int i = threadId; i < nunique; i += blockDim.x) {
        local_histogram[i] = 0;
    }
    __syncthreads();

    // Each thread processes a chunk of data and updates the local histogram
    for (int i = 0; i < chunk_size; i++) {
        int dataIdx = globalId * chunk_size + i;
        if (dataIdx < n) {
            atomicAdd(&local_histogram[data[dataIdx]], 1);
        }
    }
    __syncthreads();

    // Update global histogram from local histograms
    for (int i = threadId; i < nunique; i += blockDim.x) {
        atomicAdd(&global_histogram[i], local_histogram[i]);
    }
}
*/

__global__ void count_occurrences_kernel(int* data, int* histogram, int n, int nunique, int chunk_size) {
    int globalId = threadIdx.x + blockIdx.x * blockDim.x;

    // Each thread processes a chunk of data and updates the histogram directly
    for (int i = 0; i < chunk_size; i++) {
        int dataIdx = globalId * chunk_size + i;
        if (dataIdx < n) {
            atomicAdd(&histogram[data[dataIdx]], 1);
        }
    }
}

__global__ void histogram_to_binary(int* histogram, int* binary, int nunique) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nunique) {
        binary[index] = (histogram[index] == 1) ? 1 : 0;
    }
}

__global__ void simple_prefix_sum(int* input, int* output, int n) {
    extern __shared__ int temp[];

    int threadId = threadIdx.x;

    // Load input into shared memory.
    temp[threadId] = (threadId < n) ? input[threadId] : 0;
    __syncthreads();

    for (int stride = 1; stride < n; stride *= 2) {
        int value = 0;
        if (threadId >= stride) {
            value = temp[threadId - stride];
        }
        __syncthreads();
        temp[threadId] += value;
        __syncthreads();
    }

    if (threadId < n) {
        output[threadId] = temp[threadId];
    }
}

__global__ void extract_unique_values(int* histogram, int* prefixSum, int* unique_values, int nunique) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nunique) {
        if (histogram[index] == 1) {
            unique_values[prefixSum[index] - 1] = index;
        }
    }
}

UniqueFinder::UniqueFinder(const std::vector<int>& data, int nunique) : data(data), unique_values(nunique) {}

std::vector<int> UniqueFinder::find_unique() {
    int n = this->data.size();
    int nunique = this->unique_values;

    int* d_data;
    int* d_histogram;
    hipError_t err;

    err = hipMalloc(&d_data, n * sizeof(int));
    hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device memory: " << hipGetErrorString(err) << std::endl;
    }
    hipMalloc(&d_histogram, nunique * sizeof(int));
    hipDeviceSynchronize();
    err = hipMemcpy(d_data, data.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Failed to copy to a device memory: " << hipGetErrorString(err) << std::endl;
    }
    hipMemset(d_histogram, 0, nunique * sizeof(int));
    hipDeviceSynchronize();

    // Obtain the histogram of the data
    int blocks_count = (n + CHUNK_SIZE - 1) / CHUNK_SIZE;
    count_occurrences_kernel<<<blocks_count, BLOCK_SIZE, nunique * sizeof(int)>>>(d_data, d_histogram, n, nunique, CHUNK_SIZE);
    hipDeviceSynchronize();
    // After generating the histogram
    int* h_histogram_debug = new int[nunique];
    hipMemcpy(h_histogram_debug, d_histogram, nunique * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < nunique; i++) {
        std::cout << "Hist[" << i << "]: " << h_histogram_debug[i] << std::endl;
    }
    delete[] h_histogram_debug;

    // Convert histogram to binary format
    int* d_binary;
    hipMalloc(&d_binary, nunique * sizeof(int));
    histogram_to_binary<<<(nunique + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_histogram, d_binary, nunique);

    // After converting to binary
    int* h_binary_debug = new int[nunique];
    hipMemcpy(h_binary_debug, d_binary, nunique * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < nunique; i++) {
        std::cout << "Binary[" << i << "]: " << h_binary_debug[i] << std::endl;
    }
    delete[] h_binary_debug;

    // Allocate memory for prefix_sum and unique_values on the device
    int* d_prefix_sum, *d_unique_values;
    hipMalloc(&d_prefix_sum, nunique * sizeof(int));
    hipMalloc(&d_unique_values, nunique * sizeof(int));

    // Compute prefix sum
    int blockSize = min(nunique, BLOCK_SIZE);
    simple_prefix_sum<<<1, blockSize, blockSize * sizeof(int)>>>(d_binary, d_prefix_sum, nunique);

    // After computing prefix sum
    int* h_prefix_sum_debug = new int[nunique];
    hipMemcpy(h_prefix_sum_debug, d_prefix_sum, nunique * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < nunique; i++) {
        std::cout << "PrefixSum[" << i << "]: " << h_prefix_sum_debug[i] << std::endl;
    }
    delete[] h_prefix_sum_debug;

    // Extract unique values based on the prefix sum
    extract_unique_values<<<(nunique + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_histogram, d_prefix_sum, d_unique_values, nunique);

    // 1. Get the number of unique values
    int num_unique;
    hipMemcpy(&num_unique, &d_prefix_sum[nunique - 1], sizeof(int), hipMemcpyDeviceToHost);

    // 2. Allocate space for these unique values on the host
    std::vector<int> unique_elements(num_unique);

    // 3. Copy the unique values from the device to the host memory
    hipMemcpy(unique_elements.data(), d_unique_values, num_unique * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_data);
    hipFree(d_histogram);
    hipFree(d_prefix_sum);
    hipFree(d_binary);
    hipFree(d_unique_values);

    return unique_elements;
}
